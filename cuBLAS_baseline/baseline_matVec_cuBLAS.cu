#include <cassert>
#include <cstdint>
#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cstring>

#include <sys/time.h>
#include <fstream>
#include <vector>

#define EPSILON 1e-5 
#define MAX_BLOCKS 256
#define MAX_THREADS 256

// #define VERBOSE

bool validateResults(float* hostMat, float* hostVec, float* hostResVec, int w, int h);
void checkCUDAError(const char* msg);

class Matrix{
public:
    uint32_t rows;
    uint32_t cols;
    float* data;

    Matrix(uint32_t r, uint32_t c, float* d) : rows(r), cols(c), data(d) {}

    __host__ static Matrix deserialize(std::ifstream& file){
        uint32_t rows, cols;
        float grid_spacing;

        file.read(reinterpret_cast<char*>(&rows), sizeof(rows));
        file.read(reinterpret_cast<char*>(&cols), sizeof(cols));

        file.read(reinterpret_cast<char*>(&grid_spacing), sizeof(float));
        
        int8_t*  int_data = new int8_t[rows*cols];
        float* data; 
        printf("rows: %d, cols: %d\n",rows, cols);
        hipHostAlloc(&data, rows*cols*sizeof(float), hipHostMallocDefault);
        file.read(reinterpret_cast<char*>(int_data), rows*cols*sizeof(int8_t));

        
        // Casting to float here, watchout!
        for (int i = 0; i < rows*cols; ++i) {
                data[i] = static_cast<float>(int_data[i]);
        }

        if ((cols * rows) % 2 == 1){
            file.seekg(1, std::ios::cur);
            printf("Odd number of Matrix elements: skipped padding.");
        }

        delete [] int_data;

        return Matrix(rows, cols, data);
    }
    
    float* mult(hipblasHandle_t handle, float* result, float* vector);
};
float* Matrix::mult(hipblasHandle_t handle, float* result, float* vector){
    float alpha = 1.0f;  // Scalar multiplier for matrix-vector product
    float beta = 0.0f;   // Scalar multiplier for the initial value of y (should be 0 if we're just doing the product)
    int lda = this->rows;         // Leading dimension of matrix A
    int incx = 1;        // Increment for vector x
    int incy = 1;        // Increment for vector y
    int num_elems = this->cols*this->rows;


    float* tmp = new float[num_elems];

    hipMemcpy(tmp,this->data, sizeof(float)*lda, hipMemcpyDeviceToHost);
    printf("#+#+#+#+#");
    printf("%f\n",tmp[0]);

    hipblasStatus_t stat;
    checkCUDAError("Before Sgemv");
    stat = hipblasSgemv(handle, HIPBLAS_OP_N, this->rows, this->cols, &alpha, this->data, lda, vector, incx, &beta, result, incy);
    checkCUDAError("after Sgemv");
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("ERROR CUBLAS_STATUS_SUCCESS)");
    }

    hipMemcpy(tmp,result, sizeof(float)*lda, hipMemcpyDeviceToHost);
    checkCUDAError("after tmp memcpy");

    printf("####");
    printf("%f\n",tmp[0]);
    delete[] tmp;

    return result;
}

int main(int argc, char* argv[]) {

    std::string filename = "/home/wildug/RSP/myKernel/raw-matrices.bin";
    std::ifstream file(filename, std::ios::binary);
    if (!file) {
        std::cerr << "Error: Could not open file" << std::endl;
        return 1;
    }


    // Initialize cuBLAS handle
    float ms = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    uint32_t num_matrices, len_v;
    int8_t* int_vec;

    file.read(reinterpret_cast<char*>(&num_matrices), sizeof(num_matrices));
    file.read(reinterpret_cast<char*>(&len_v), sizeof(len_v));

    int_vec = new int8_t[len_v];
    std::cout << "Number of matrices: " << num_matrices << std::endl;
    std::cout << "len_v: " << len_v << std::endl;
    file.read(reinterpret_cast<char*>(int_vec), len_v*sizeof(uint8_t));

    float* h_vec = new float[len_v];
    float* vec;


    for (int i = 0; i < len_v; ++i) {
            h_vec[i] = static_cast<float>(int_vec[i]);
            printf("%f, ",h_vec[i]);
    }
    printf("end\n");
    hipMalloc(&vec, sizeof(float)*len_v);
    hipMemcpy(vec, h_vec, sizeof(float)*len_v,hipMemcpyHostToDevice);



    checkCUDAError("after Reading");

    
    std::vector<Matrix> matrices;
    for (int k = 0; k<num_matrices; k++){
        printf("%d\n",k);
        Matrix matrix = Matrix::deserialize(file);
        matrices.push_back(std::move(matrix));
    }

    file.close();
    for (int k = 0; k<num_matrices; k++){
        Matrix& matrix = matrices[k];
        float* d_data;
        int num_elems = matrix.rows*matrix.cols;

        printf("%d,",k);
        printf("elms: %d, ",matrix.cols*matrix.rows);
        checkCUDAError("before Malloc");
        hipMalloc((void**)&d_data, sizeof(float)*num_elems);
        checkCUDAError("after Malloc");
        hipMemcpy(d_data, matrix.data,sizeof(float)*num_elems, hipMemcpyHostToDevice);
        checkCUDAError("after Memcpy");


        matrix.data = d_data;
    }

    checkCUDAError("after loop");
    int max_rows = 0;
    for (const auto& matrix : matrices) {
        if (matrix.rows > max_rows)
            max_rows = matrix.rows;
    }

    printf("max_rows: %d", max_rows);

    // cudaMalloc(&d_result, sizeof(float)*max_rows);

    checkCUDAError("after allocating d_result");
    int rows;
    for (int k = 0; k<num_matrices; k++){
        Matrix matrix = matrices[k];
        rows = matrix.rows;
        float* d_result;
        hipMalloc(&d_result, sizeof(float)* rows); //TODO allocate outside the loop
        matrix.mult(handle, d_result, vec);

        checkCUDAError("after multiplying matrix");
        vec = d_result;
    }
    
    hipMemcpy(h_vec, vec, sizeof(float)*rows, hipMemcpyDeviceToHost);
    hipFree(vec);
    for (int k = 0; k<num_matrices; k++){
        Matrix matrix = matrices[k];
        hipFree(matrix.data);
    }

//     // Perform matrix-vector multiplication y = alpha * A * x + beta * y

//     cudaEventRecord(start);
//     // d_a is stored in column MAJOR

//     // timing
//     cudaEventRecord(stop);
//     cudaEventSynchronize(stop);
//     cudaEventElapsedTime(&ms, start, stop);
//     printf("Kernel execution time: %.6f ms\n", ms);


    // Output result
    std::cout << "Result vector y: ";
    for (int i = 0; i < rows; i++) {
        std::cout << h_vec[i] << " ";
    }
    std::cout << std::endl;



    // // Cleanup
    // cudaFree(d_A);
    // cudaFree(d_x);
    // cudaFree(d_y);

    // Destroy cuBLAS handle
    hipblasDestroy(handle);

    return 0;
}

bool validateResults(float* hostMat, float* hostVec, float* gpuResult, int w, int h) {

    float* result = new float[h];

    for (int i = 0; i < h; ++i) {
        result[i] = 0.0f;
        for (int j = 0; j < w; ++j) {
            result[i] += hostMat[i * w + j] * hostVec[j];
        }
    }
    for (int i = 0; i < h; ++i) {
        if (fabs(result[i] - gpuResult[i]) > EPSILON) {
            printf("Mismatch at index %d: CPU=%f, GPU=%f\n", i, result[i], gpuResult[i]);
            return false;
        }
    }
    delete [] result;
    return true;
}

void checkCUDAError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}