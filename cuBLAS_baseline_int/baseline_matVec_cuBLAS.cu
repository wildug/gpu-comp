#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdint>
#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <fstream>
#include <vector>

#include <hipblaslt.h>
#include <hip/hip_runtime_api.h>

#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <cmath>

#define EPSILON 1e-5 
#define MAX_BLOCKS 256
#define MAX_THREADS 256

#define VERBOSE

bool validateResults(float* hostMat, float* hostVec, float* hostResVec, int w, int h);
void checkCUDAError(const char* msg);

inline void checkCublasStatus(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS API failed with status %s\n", cublasLtGetStatusString(status));
        throw std::logic_error("cuBLAS API failed");
    }
}

struct AbsValue {
    __host__ __device__
    float operator()(const int32_t& x) const {
        return abs(x);
    }
};

float absMaxWithThrustDevice(int32_t* d_input, int n) {
    thrust::device_ptr<int32_t> dev_ptr(d_input);

    return thrust::transform_reduce(
        dev_ptr, dev_ptr + n,
        AbsValue(),              // transform: fabs(x)
        0.0f,                    // init
        thrust::maximum<int32_t>() // reduce: max
    );
}

__global__ void normalizeAndRoundtoInt8(int32_t* res32, int8_t* res8, float scalar, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        int32_t a = res32[idx];
        float afl = static_cast<float>(a);

        a = __float2int_rn(afl/ scalar);
        res8[idx] = static_cast<int8_t>(a);
    }
}


class Matrix{
public:
    uint32_t rows;
    uint32_t cols;
    float w_delta;
    int8_t* data;

    Matrix(uint32_t r, uint32_t c, float w_delta, int8_t* d) : rows(r), cols(c), w_delta(w_delta), data(d) {}

    __host__ static Matrix deserialize(std::ifstream& file){
        uint32_t rows, cols;
        float grid_spacing;

        file.read(reinterpret_cast<char*>(&rows), sizeof(rows));
        file.read(reinterpret_cast<char*>(&cols), sizeof(cols));

        file.read(reinterpret_cast<char*>(&grid_spacing), sizeof(float));
        
        int8_t*  int_data;
        hipHostMalloc(&int_data, rows*cols*sizeof(int8_t)) ;
        printf("rows: %d, cols: %d\n",rows, cols);
        file.read(reinterpret_cast<char*>(int_data), rows*cols*sizeof(int8_t));

        

        if ((cols * rows) % 2 == 1){
            file.seekg(1, std::ios::cur);
            printf("Odd number of Matrix elements: skipped padding.");
        }

        return Matrix(rows, cols, grid_spacing, int_data);
    }
    
        float mult(hipblasLtHandle_t ltHandle, int32_t* d_result32, int8_t* result, int8_t* vector, float v_delta, void *workspace, hipblasLtMatmulAlgo_t algo);
};

float Matrix::mult(hipblasLtHandle_t ltHandle, int32_t* d_result32, int8_t* result, int8_t* vector, float v_delta, void *workspace, hipblasLtMatmulAlgo_t algo){
    // returns float value 

    int32_t alpha = 1;  // Scale of vector and matrix quantization
    int32_t beta = 0.0f;   // Scalar multiplier for the initial value of y (should be 0 if we're just doing the product)
    int rows = this->rows;         // Leading dimension of matrix A
    int cols = this->cols;
    int num_elems = this->cols*this->rows;



    checkCUDAError("Before Sgemv");

    hipblasLtMatrixLayout_t Adesc, Bdesc, Ddesc;
    hipblasLtMatmulDesc_t matmulDesc;

    int m = rows, n = 1, k = cols;
    int lda = rows, ldb = cols, ldc = rows;
    hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_8I, m, k, lda);
    hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_8I, k, n, ldb);
    hipblasLtMatrixLayoutCreate(&Ddesc, HIP_R_32I, m, n, ldc);
    checkCublasStatus(hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32I, HIP_R_32I));
 

    checkCublasStatus(
    hipblasLtMatmul(
            ltHandle,
            matmulDesc,
            &alpha,
            this->data, Adesc,
            vector, Bdesc,
            &beta,
            d_result32, Ddesc,
            d_result32, Ddesc,
            &algo,
            workspace, 1<<20,
            0
        ));
    
    checkCUDAError("after Sgemv");
    float abs_max = absMaxWithThrustDevice(d_result32, this->rows);

    v_delta = abs_max / 127;

    int blocks = (rows+ MAX_THREADS - 1) / MAX_THREADS;

    normalizeAndRoundtoInt8<<< blocks,MAX_THREADS>>>
    (d_result32, result, v_delta, this->rows);
    checkCUDAError("after normalizeAndRound");


    return v_delta;
}
int main(int argc, char* argv[]) {

    std::string filename = "/home/wildug/RSP/myKernel/raw-matrices.bin";

    // std::string filename = "/home/ludwigal/readMat/compressed_matrices.bin";
    std::ifstream file(filename, std::ios::binary);
    if (!file) {
        std::cerr << "Error: Could not open file" << std::endl;
        return 1;
    }


    // Initialize cuBLAS handle
    float ms = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // cuBLASLt handle
    hipblasLtHandle_t ltHandle;
    hipblasLtCreate(&ltHandle);

    uint32_t num_matrices, len_v;
    int8_t* h_vec;

    file.read(reinterpret_cast<char*>(&num_matrices), sizeof(num_matrices));
    file.read(reinterpret_cast<char*>(&len_v), sizeof(len_v));

    h_vec = new int8_t[len_v];
    std::cout << "Number of matrices: " << num_matrices << std::endl;
    std::cout << "len_v: " << len_v << std::endl;
    file.read(reinterpret_cast<char*>(h_vec), len_v*sizeof(uint8_t));

    int8_t* vec;


    printf("end\n");
    hipMalloc(&vec, sizeof(int8_t)*len_v);
    hipMemcpy(vec, h_vec, sizeof(int8_t)*len_v,hipMemcpyHostToDevice);

    checkCUDAError("after Reading");

    std::vector<Matrix> matrices;
    for (int k = 0; k<num_matrices; k++){
        printf("%d\n",k);
        Matrix matrix = Matrix::deserialize(file);
        matrices.push_back(std::move(matrix));
    }

    file.close();
    hipEventRecord(start);

    for (int k = 0; k<num_matrices; k++){
        Matrix& matrix = matrices[k];
        int8_t* d_data;
        int num_elems = matrix.rows*matrix.cols;

        checkCUDAError("before Malloc");
        hipMalloc(&d_data, sizeof(int8_t)*num_elems);
        checkCUDAError("after Malloc");
        hipMemcpy(d_data, matrix.data,sizeof(int8_t)*num_elems, hipMemcpyHostToDevice);
        checkCUDAError("after Memcpy");


        matrix.data = d_data;
    }


    checkCUDAError("after loop");
    int max_rows = 0;
    for (const auto& matrix : matrices) {
        if (matrix.rows > max_rows)
            max_rows = matrix.rows;
    }

    printf("max_rows: %d\n", max_rows);

    int32_t* d_result32;
    int8_t* d_result8;
    int8_t* blob;
    hipMalloc(&d_result32, sizeof(int32_t)*max_rows);
    hipMalloc(&d_result8, sizeof(int8_t)*max_rows);

    checkCUDAError("after allocating d_result");
    int rows;
    int v_delta = 1; // scaling factor of v starts with 1

    // cuBLASlt
    int m = 4096, n = 1, k = 4096;
    int lda = 4096, ldb = 4096, ldc = 4096;

    hipblasLtMatrixLayout_t Adesc, Bdesc, Cdesc;
    hipblasLtMatmulDesc_t matmulDesc;

    // A: m x k, B: k x n, C: m x n
    hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_8I, m, k, lda);
    hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_8I, k, n, ldb);
    hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32I, m, n, ldc);
    hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32I, HIP_R_32I);
    // Heuristic algorithm search
    hipblasLtMatmulAlgo_t algo;
    hipblasLtMatmulPreference_t preference;
    hipblasLtMatmulPreferenceCreate(&preference);
    void *workspace = nullptr;
    size_t workspaceSize = 1 << 22; // arbitrary

    hipMalloc(&workspace, workspaceSize);
    hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));

    int returnedResults = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult;


    hipblasLtMatmulAlgoGetHeuristic(ltHandle, matmulDesc, Adesc, Bdesc, Cdesc, Cdesc, preference, 1, &heuristicResult, &returnedResults);
    algo = heuristicResult.algo;

    for (int k = 0; k<num_matrices; k++){
        Matrix matrix = matrices[k];
        rows = matrix.rows;
        v_delta = matrix.mult(ltHandle, d_result32, d_result8, vec, v_delta, workspace, algo);
        checkCUDAError("after multiplying matrix");
        blob = vec;
        vec = d_result8;
        d_result8 = blob;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf("%f ms\n", ms);
    checkCUDAError("after loop");
    
    hipMemcpy(h_vec, vec, sizeof(int8_t)*rows, hipMemcpyDeviceToHost);
    hipFree(vec);
    for (int k = 0; k<num_matrices; k++){
        Matrix matrix = matrices[k];
        hipFree(matrix.data);
    }




    // Output result
    printf("[");
    for (int i=0; i<rows; i++){
        // printf("Result at index %d: %d\n", i, h_result[i]);
        printf("%d,",  h_vec[i]);
    }
    printf("]\n");



    hipblasLtDestroy(ltHandle);

    hipDeviceReset();
    checkCUDAError("End of program.");

    return 0;
}

bool validateResults(float* hostMat, float* hostVec, float* gpuResult, int w, int h) {

    float* result = new float[h];

    for (int i = 0; i < h; ++i) {
        result[i] = 0.0f;
        for (int j = 0; j < w; ++j) {
            result[i] += hostMat[i * w + j] * hostVec[j];
        }
    }
    for (int i = 0; i < h; ++i) {
        if (fabs(result[i] - gpuResult[i]) > EPSILON) {
            printf("Mismatch at index %d: CPU=%f, GPU=%f\n", i, result[i], gpuResult[i]);
            return false;
        }
    }
    delete [] result;
    return true;
}

void checkCUDAError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}