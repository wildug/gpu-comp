#include "hip/hip_runtime.h"
#include <cassert>
#include <cstddef>
#include <cstdio>
#include <fstream>
#include <cstdint>
#include <cstring>
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>

#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>

#define MAX_BLOCKS 256
#define MAX_THREADS 256

void checkCUDAError(const char* msg);

__inline__ __device__ int8_t find_r(uint8_t quantile, uint8_t* cdf, int G){

    for (int8_t r=G; r>0; r--){
        if (cdf[r-1] <= quantile){
            return r-1;
        }
    }
    return -1;
};

struct AbsValue {
    __host__ __device__
    float operator()(const int32_t& x) const {
        return abs(x);
    }
};

float absMaxWithThrustDevice(int32_t* d_input, int n) {
    thrust::device_ptr<int32_t> dev_ptr(d_input);

    return thrust::transform_reduce(
        dev_ptr, dev_ptr + n,
        AbsValue(),              // transform: fabs(x)
        0.0f,                    // init
        thrust::maximum<int32_t>() // reduce: max
    );
}

__global__ void normalizeAndRoundtoInt8(int32_t* res32, int8_t* res8, float scalar, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        int32_t a = res32[idx];
        float afl = static_cast<float>(a);

        a = __float2int_rn(afl/ scalar);
        res8[idx] = static_cast<int8_t>(a);
    }
}


class CompressedMatrix {
public:
    uint32_t rows, cols;
    float grid_spacing;
    int8_t min_value;
    uint8_t G;
    // host array pointers
    uint32_t* cursors;
    uint8_t* cdf_data;
    uint8_t* ppf_data;
    uint16_t* payload;
    uint32_t payload_size;

    // device array pointers
    uint32_t* d_cursors;
    uint8_t* d_cdf_data;
    uint8_t* d_ppf_data;
    uint16_t* d_payload;

    CompressedMatrix(uint32_t r, uint32_t c, float gs, uint32_t* cur,
                      int8_t minVal, uint8_t G, uint8_t* cdf, uint8_t* ppf, uint16_t* pay, uint32_t pay_size)
        : rows(r), cols(c), grid_spacing(gs), cursors(cur), min_value(minVal), G(G),
          cdf_data(cdf), ppf_data(ppf), payload(pay), payload_size(pay_size), d_cursors(nullptr), d_cdf_data(nullptr),
           d_ppf_data(nullptr), d_payload(nullptr){}

    __host__ static CompressedMatrix deserialize(std::ifstream& file) {
        uint32_t rows, cols;
        float grid_spacing;

        uint32_t payload_size;
        int8_t min_value;
        uint8_t G;
        file.read(reinterpret_cast<char*>(&rows), sizeof(rows));
        file.read(reinterpret_cast<char*>(&cols), sizeof(cols));
        file.read(reinterpret_cast<char*>(&grid_spacing), sizeof(float));
        
        // uint32_t* cursors = new uint32_t[rows];

        // non-pageable memoryP
        uint32_t* cursors;
        hipHostMalloc(&cursors, sizeof(uint32_t)*rows);

        file.read(reinterpret_cast<char*>(cursors), rows * sizeof(uint32_t));

        file.read(reinterpret_cast<char*>(&payload_size), sizeof(payload_size));

        file.read(reinterpret_cast<char*>(&min_value), sizeof(int8_t));
        file.read(reinterpret_cast<char*>(&G), sizeof(uint8_t));

        uint32_t cdf_len = G + 1;
        // uint8_t* cdf_data = new uint8_t[cdf_len];
        uint8_t* cdf_data;
        hipHostMalloc(&cdf_data, sizeof(uint8_t)*cdf_len);
        file.read(reinterpret_cast<char*>(cdf_data), cdf_len);
        
        if (cdf_len % 2 == 1) {
            file.seekg(1, std::ios::cur);
        }

        // uint8_t* ppf_data = new uint8_t[256];;
        uint8_t* ppf_data;
        hipHostMalloc(&ppf_data, sizeof(uint8_t)*256);
        file.read( reinterpret_cast<char*>(ppf_data), 256);
        
        // uint16_t* payload = new uint16_t[payload_size];
        uint16_t* payload;
        hipHostMalloc(&payload, sizeof(uint16_t)*payload_size);
        file.read(reinterpret_cast<char*>(payload), payload_size * sizeof(uint16_t));
        
        if (payload_size % 2 == 1) {
            file.seekg(2, std::ios::cur);
        }

        return CompressedMatrix(rows, cols, grid_spacing, cursors, min_value, G, cdf_data, ppf_data, payload, payload_size);
    }

    float decompressAndMult(int8_t* result, int32_t* d_result32, int8_t* vector, float v_delta);
};

__global__ void decmpressAndMultiply(int32_t* dst, int8_t* vec,
     uint32_t rows, uint32_t cols, float grid_spacing,
     uint32_t* cursors, int8_t min_value, uint8_t G,
     uint8_t* cdf_data, uint8_t* ppf_data, uint16_t* payload, uint32_t payload_size
){
    unsigned int threadNo = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int tId = threadIdx.x;
    // unsigned int bId = blockIdx.x;
    unsigned int blockSize = blockDim.x;
    
    
    uint32_t cursor;
    uint32_t head;
    uint8_t quantile;
    uint8_t r;
    int8_t w;
    uint8_t prob;

    __shared__ int8_t shared_vec[4096]; // TODO NOT HARDCODE THIS NUMBER
    extern __shared__ uint8_t cdf[]; // store cdf in shared memory
    __shared__ uint8_t ppf[256];


    for (int j = tId; j <cols; j+=blockSize ){
        shared_vec[j] = vec[j];
    }

    int32_t res = 0;

    // loads cdf & ppf into shared memory 
    for (int j = tId; j <G+1; j+=blockSize ){
        cdf[j] = cdf_data[j];
    }
    for (int j=tId; j< 256; j+=blockSize){
        ppf[j] = ppf_data[j];
    }

    __syncthreads();
    
    if (threadNo < rows){
        cursor = cursors[threadNo];
        head = payload[cursor] << 16 | payload[cursor+1];
        cursor +=2;
        for (int j = 0; j < cols; j++){
            quantile = head & ((1<<8)-1); // take first 8 bits of head as quantile

            r = ppf[quantile];
            // r = find_r(quantile, cdf, G);


            w = min_value + r;

            // __dp4a(srcA, srcB,c); 
            // see https://docs.nvidia.com/cuda/cuda-math-api/cuda_math_api/group__CUDA__MATH__INTRINSIC__INT.html#group__cuda__math__intrinsic__int_1ga933213059df6da2de206771f145ac2f8


            res += w * shared_vec[j]; // perform scalar addition

            prob = (cdf[r+1] - cdf[r]) & 0xFF; // modulo 2**8 to ensure it fits in a uint8
            head = (head >> 8) * prob  + (quantile -cdf[r]);
            if (head < (1<<16)){
                head = head<<16 | payload[cursor];
                cursor+=1;
            }
        }
        dst[threadNo] = res;
    }   
}
float CompressedMatrix::decompressAndMult(int8_t* d_result8, int32_t* d_result32, int8_t* vector, float v_delta){

    dim3 blockGrid(MAX_BLOCKS);
    dim3 threadBlock(MAX_THREADS);
    
    

    decmpressAndMultiply<<<blockGrid, threadBlock, (G+1)*sizeof(int8_t)>>>(d_result32, vector,
        this->rows, this->cols, this->grid_spacing,
        this-> d_cursors, this->min_value, this->G,
        this->d_cdf_data,this->d_ppf_data, this->d_payload, this->payload_size);


    float abs_max = absMaxWithThrustDevice(d_result32, this->rows);
    v_delta = abs_max / 127;
    
    int blocks = (rows+ MAX_THREADS - 1) / MAX_THREADS;
    normalizeAndRoundtoInt8<<< blocks,MAX_THREADS>>>
    (d_result32, d_result8, v_delta, this->rows);

    checkCUDAError("after kernel");
    return v_delta;
}


int main() {
    // Open the binary file
    std::string filename = "/home/wildug/RSP/myKernel/compressed_matrices.bin";
    // std::string filename = "/home/wildug/Downloads/compressed_matrices.bin";
    std::ifstream file(filename, std::ios::binary);
    
    // for timing
    float ms = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    if (!file) {
        std::cerr << "Error: Could not open file" << std::endl;
        return 1;
    }

    uint32_t num_matrices, max_word_count, len_v;
    int8_t* v0;
    int8_t* vec;
    file.read(reinterpret_cast<char*>(&num_matrices), sizeof(num_matrices));
    file.read(reinterpret_cast<char*>(&max_word_count), sizeof(max_word_count));
    file.read(reinterpret_cast<char*>(&len_v), sizeof(len_v));


    int8_t* h_v0 = new int8_t[len_v];
    file.read(reinterpret_cast<char*>(h_v0), len_v*sizeof(uint8_t));

    hipMalloc(&v0, sizeof(int8_t)*len_v);
    hipMemcpy(v0, h_v0,sizeof(int8_t)*len_v,  hipMemcpyHostToDevice);

    hipMalloc(&vec,sizeof(int8_t)*len_v);

    std::cout << "Number of matrices: " << num_matrices << std::endl;
    std::cout << "Max word-count: " << max_word_count << std::endl;
    std::cout << "len_v: " << len_v << std::endl;



    // maybe first read all files and then do the mat vec operation
    int8_t* d_result;
    int8_t* h_result;
    int rows =len_v;
    int8_t* tmp;
    std::vector<CompressedMatrix> encoded_matrices;

    for (int k = 0; k<num_matrices; k++){
        CompressedMatrix matrix = CompressedMatrix::deserialize(file);
        encoded_matrices.push_back(std::move(matrix));
    }


    file.close();

    h_result = new int8_t[rows];
    

    int max_rows;
    for (int k = 0; k<num_matrices; k++){
        if (max_rows < encoded_matrices[k].rows)              
            max_rows = encoded_matrices[k].rows;
    }


    hipMalloc(&d_result, sizeof(int8_t)* max_rows); 

    int32_t* d_result32;
    hipMalloc(&d_result32, sizeof(int32_t)* max_rows); 
    
    // alternatively put this inside benchmarking loop
    // MEMCPY LOOP, move hipEventRecord above or below
    for (int k = 0; k<num_matrices; k++){
        printf("%d\n",k);
        CompressedMatrix& matrix = encoded_matrices[k];
        uint32_t* d_cursors;
        uint8_t* d_cdf_data;
        uint8_t* d_ppf_data;
        uint16_t* d_payload;

        // malloc
        checkCUDAError("before Malloc");
        hipMalloc(&d_cursors, sizeof(uint32_t)* matrix.rows);
        hipMalloc(&d_cdf_data, sizeof(uint8_t)*(matrix.G +1));
        hipMalloc(&d_ppf_data, 256*sizeof(uint8_t));
        hipMalloc(&d_payload, matrix.payload_size * sizeof(uint16_t));

        checkCUDAError("after Malloc");

        // memcpy
        hipMemcpy(d_cursors, matrix.cursors, sizeof(uint32_t)*matrix.rows, hipMemcpyHostToDevice);
        hipMemcpy(d_cdf_data, matrix.cdf_data,sizeof(uint8_t)*(matrix.G +1), hipMemcpyHostToDevice);
        hipMemcpy(d_ppf_data, matrix.ppf_data,sizeof(uint8_t)*256, hipMemcpyHostToDevice);
        hipMemcpy(d_payload, matrix.payload, matrix.payload_size * sizeof(uint16_t), hipMemcpyHostToDevice);

        checkCUDAError("after Memcpy");


        // set the *device* pointer as object attribute
        matrix.d_cursors = d_cursors;
        matrix.d_cdf_data = d_cdf_data;
        matrix.d_ppf_data = d_ppf_data;
        matrix.d_payload =  d_payload;
    }

    for (int l=0; l< 10; l++){ // outer loop for benchmarking

        hipMemcpy(vec, v0, sizeof(int8_t)*len_v, hipMemcpyDeviceToDevice);

        hipEventRecord(start);

        float v_delta = 1;

        // COMPUTE LOOP
        for (int k = 0; k<num_matrices; k++){
            CompressedMatrix& matrix = encoded_matrices[k];

            matrix.decompressAndMult(d_result, d_result32, vec, v_delta);
            checkCUDAError("after decompressing matrix");

            // to swap variables you need a third guy 'tmp'
            tmp = vec;
            vec = d_result;
            d_result = tmp;


            checkCUDAError("sizes misalign");
            rows = matrix.rows;
        }
        hipDeviceSynchronize();



        checkCUDAError("Before Memcpy.");

        // copy 'vec' since we swapped it with d_result
        hipMemcpy(h_result, vec, sizeof(int8_t)* rows, hipMemcpyDeviceToHost);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms, start, stop);
        printf("%f ms\n", ms);
        
        // alternatively put this outside of benchmarking loop
    }

    printf("freeing Memory: ");
    for (int k = 0; k<num_matrices; k++){
        CompressedMatrix matrix = encoded_matrices[k];

        hipHostFree(matrix.cursors);
        hipHostFree(matrix.cdf_data);
        hipHostFree(matrix.ppf_data);
        hipHostFree(matrix.payload);
        hipFree(matrix.d_cursors);
        hipFree(matrix.d_cdf_data);
        hipFree(matrix.d_ppf_data);
        hipFree(matrix.d_payload);
    }
    // show result
    printf("[");
    for (int i=0; i<rows; i++){
        // printf("Result at index %d: %d\n", i, h_result[i]);
        printf("%d,",  h_result[i]);
    }
    printf("]\n");
    hipFree(d_result);
    delete[] h_v0;
    delete[] h_result;


    hipDeviceReset();
    checkCUDAError("End of program.");
    
    return 0;
}

void checkCUDAError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}