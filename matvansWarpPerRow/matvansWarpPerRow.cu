#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdio>
#include <fstream>
#include <cstdint>
#include <cstring>
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include "matvansWarpPerRow-kernel.cu"

#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>

#define MAX_BLOCKS 256
#define MAX_THREADS 256
#define WARP_SIZE 32

void checkCUDAError(const char* msg);

__inline__ __device__ int8_t find_r(uint8_t quantile, uint8_t* cdf, int G){

    for (int8_t r=G; r>0; r--){
        if (cdf[r-1] <= quantile){
            return r-1;
        }
    }
    return -1;
};

uint32_t hash_int8_array(int8_t* arr, int size)
{
    uint32_t hash = 0;

    for (size_t i = 0; i < size; i++)
    {
        hash = (hash >> 27) | (hash << 5); // Rotate left by 5 bits
        hash = (hash ^ *reinterpret_cast<const uint8_t *>(&arr[i])) * 0x27220A95;
    }

    return hash;
}

struct AbsValue {
    __host__ __device__
    float operator()(const int32_t& x) const {
        return abs(x);
    }
};

float absMaxWithThrustDevice(int32_t* d_input, int n) {
    thrust::device_ptr<int32_t> dev_ptr(d_input);

    return thrust::transform_reduce(
        dev_ptr, dev_ptr + n,
        AbsValue(),              // transform: fabs(x)
        0.0f,                    // init
        thrust::maximum<int32_t>() // reduce: max
    );
}

__global__ void normalizeAndRoundtoInt8(int32_t* res32, int8_t* res8, float scalar, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        int32_t a = res32[idx];
        float afl = static_cast<float>(a);

        a = __float2int_rn(afl/ scalar);
        res8[idx] = static_cast<int8_t>(a);
    }
}


class CompressedMatrix {
public:
    uint32_t rows, cols;
    float grid_spacing;
    int8_t min_value;
    uint8_t G;
    // host array pointers
    uint32_t* cursors;
    uint8_t* cdf_data;
    uint8_t* ppf_data;
    uint16_t* payload;
    uint32_t payload_size;

    // device array pointers
    uint32_t* d_cursors;
    uint8_t* d_cdf_data;
    uint8_t* d_ppf_data;
    uint16_t* d_payload;

    CompressedMatrix(uint32_t r, uint32_t c, float gs, uint32_t* cur,
                      int8_t minVal, uint8_t G, uint8_t* cdf, uint8_t* ppf, uint16_t* pay, uint32_t pay_size)
        : rows(r), cols(c), grid_spacing(gs), cursors(cur), min_value(minVal), G(G),
          cdf_data(cdf), ppf_data(ppf), payload(pay), payload_size(pay_size), d_cursors(nullptr), d_cdf_data(nullptr),
           d_ppf_data(nullptr), d_payload(nullptr){}

    __host__ static CompressedMatrix deserialize(std::ifstream& file) {
        uint32_t rows, cols;
        float grid_spacing;

        uint32_t payload_size;
        int8_t min_value;
        uint8_t G;
        file.read(reinterpret_cast<char*>(&rows), sizeof(rows));
        file.read(reinterpret_cast<char*>(&cols), sizeof(cols));
        file.read(reinterpret_cast<char*>(&grid_spacing), sizeof(float));
        

        // non-pageable memory
        uint32_t* cursors;
        hipHostMalloc(&cursors, sizeof(uint32_t)*rows);

        file.read(reinterpret_cast<char*>(cursors), rows * sizeof(uint32_t));

        file.read(reinterpret_cast<char*>(&payload_size), sizeof(payload_size));

        file.read(reinterpret_cast<char*>(&min_value), sizeof(int8_t));
        file.read(reinterpret_cast<char*>(&G), sizeof(uint8_t));

        uint32_t cdf_len = G + 1;
        uint8_t* cdf_data;
        hipHostMalloc(&cdf_data, sizeof(uint8_t)*cdf_len);
        file.read(reinterpret_cast<char*>(cdf_data), cdf_len);
        
        if (cdf_len % 2 == 1) {
            file.seekg(1, std::ios::cur);
        }

        uint8_t* ppf_data;
        hipHostMalloc(&ppf_data, sizeof(uint8_t)*256);
        file.read( reinterpret_cast<char*>(ppf_data), 256);
        
        __align__(16) uint16_t* payload;
        hipHostMalloc(&payload, sizeof(uint16_t)*payload_size);
        file.read(reinterpret_cast<char*>(payload), payload_size * sizeof(uint16_t));
        
        if (payload_size % 2 == 1) {
            file.seekg(2, std::ios::cur);
        }

        return CompressedMatrix(rows, cols, grid_spacing, cursors, min_value, G, cdf_data, ppf_data, payload, payload_size);
    }

    float decompressAndMult(int8_t* result, int32_t* d_result32, int8_t* vector, float v_delta);
};

float CompressedMatrix::decompressAndMult(int8_t* d_result8, int32_t* d_result32, int8_t* vector, float v_delta){

    
    int rows = this->rows;
    int cols = this->cols;
    
    int warpsPerBlock = 4;
    dim3 blockGrid( (rows + warpsPerBlock - 1) / warpsPerBlock );
    dim3 threadBlock(warpsPerBlock * 32);
    

    decmpressAndMultiply<<<blockGrid, threadBlock, cols*sizeof(uint8_t)>>>(d_result32, vector,
        this->rows, this->cols, this->grid_spacing,
        this-> d_cursors, this->min_value, this->G,
        this->d_cdf_data,this->d_ppf_data, this->d_payload, this->payload_size);


    float abs_max = absMaxWithThrustDevice(d_result32, this->rows);
    v_delta = abs_max / 127;
    
    int blocks = (rows+ MAX_THREADS - 1) / MAX_THREADS;
    normalizeAndRoundtoInt8<<< blocks,MAX_THREADS>>>
    (d_result32, d_result8, v_delta, this->rows);

    checkCUDAError("after kernel");
    return v_delta;
}


int main(int argc,char *argv[]) {
    // Open the binary file
    std::vector<std::string> filepaths;

    // Case 1: One or more arguments provided
    if (argc > 1) {
        int numberOfFiles = argc - 1;
        printf("Received %d file(s) to run matvansWarpPerRow.\n", numberOfFiles);

        // Collect file paths from command line
        for (int i = 1; i < argc; ++i) {
            filepaths.push_back(argv[i]);
        }
    }
    // Case 2: No arguments -> use defaults
    else {
        filepaths = {
            "/home/wildug/RSP/myKernel/matvansWarpPerRow/compressed_matrices_4096.bin",
            "/home/bamler/bdz937/matvansWarpPerRow/compressed_matrices_4096_5bit.bin",
            "/home/ludwigal/matvansWarpPerRow/compressed_matrices_4096_5bit.bin"
        };
        printf("No input files given, trying default paths...\n");
    }

    bool opened = false;
    std::ifstream file;
    for (const auto& path : filepaths) {
        file.open(path);
        if (file.is_open()) {
            printf("\n");
            std::cout << "Opening "<< path << std::endl;
            opened = true;
            // for timing
            // time including memcpy
            float ms1 = 0;
            hipEvent_t start1, stop1;
            hipEventCreate(&start1);
            hipEventCreate(&stop1);

            // time using on
            float ms2 = 0;
            hipEvent_t start2, stop2;
            hipEventCreate(&start2);
            hipEventCreate(&stop2);


            uint32_t num_matrices, result_hash, max_word_count, len_v;
            int8_t* v0;
            int8_t* vec;
            file.read(reinterpret_cast<char*>(&num_matrices), sizeof(num_matrices));
            file.read(reinterpret_cast<char*>(&result_hash), sizeof(result_hash));
            file.read(reinterpret_cast<char*>(&max_word_count), sizeof(max_word_count));
            file.read(reinterpret_cast<char*>(&len_v), sizeof(len_v));


            int8_t* h_v0 = new int8_t[len_v];
            file.read(reinterpret_cast<char*>(h_v0), len_v*sizeof(uint8_t));

            hipMalloc(&v0, sizeof(int8_t)*len_v);
            hipMemcpy(v0, h_v0,sizeof(int8_t)*len_v,  hipMemcpyHostToDevice);

            hipMalloc(&vec,sizeof(int8_t)*len_v);

            std::cout << "Number of matrices: " << num_matrices << std::endl;
            std::cout << "Max word-count: " << max_word_count << std::endl;
            std::cout << "len_v: " << len_v << std::endl;



            // maybe first read all files and then do the mat vec operation
            int8_t* d_result;
            int8_t* h_result;
            int rows =len_v;
            int8_t* tmp;
            std::vector<CompressedMatrix> encoded_matrices;

            for (int k = 0; k<num_matrices; k++){
                CompressedMatrix matrix = CompressedMatrix::deserialize(file);
                encoded_matrices.push_back(std::move(matrix));
            }


            file.close();

            h_result = new int8_t[rows];
            

            int max_rows = 0;
            for (int k = 0; k<num_matrices; k++){
                if (max_rows < encoded_matrices[k].rows)              
                    max_rows = encoded_matrices[k].rows;
            }


            hipMalloc(&d_result, sizeof(int8_t)* max_rows); 

            int32_t* d_result32;
            hipMalloc(&d_result32, sizeof(int32_t)* max_rows); 
            
            // MEMCPY LOOP, move hipEventRecord above or below

            int NUM_ITERATIONS = 10;
            for (int l=0; l< NUM_ITERATIONS; l++){ // outer loop for benchmarking

                hipEventRecord(start1);
                for (int k = 0; k<num_matrices; k++){
                    CompressedMatrix& matrix = encoded_matrices[k];
                    uint32_t* d_cursors;
                    uint8_t* d_cdf_data;
                    uint8_t* d_ppf_data;
                    __align__(16) uint16_t* d_payload;

                    // malloc
                    checkCUDAError("before Malloc");
                    hipMalloc(&d_cursors, sizeof(uint32_t)* matrix.rows);
                    hipMalloc(&d_cdf_data, sizeof(uint8_t)*(matrix.G +1));
                    hipMalloc(&d_ppf_data, 256*sizeof(uint8_t));
                    hipMalloc(&d_payload, matrix.payload_size * sizeof(uint16_t));

                    checkCUDAError("after Malloc");

                    // memcpy
                    hipMemcpy(d_cursors, matrix.cursors, sizeof(uint32_t)*matrix.rows, hipMemcpyHostToDevice);
                    hipMemcpy(d_cdf_data, matrix.cdf_data,sizeof(uint8_t)*(matrix.G +1), hipMemcpyHostToDevice);
                    hipMemcpy(d_ppf_data, matrix.ppf_data,sizeof(uint8_t)*256, hipMemcpyHostToDevice);
                    hipMemcpy(d_payload, matrix.payload, matrix.payload_size * sizeof(uint16_t), hipMemcpyHostToDevice);

                    checkCUDAError("after Memcpy");


                    // set the *device* pointer as object attribute
                    matrix.d_cursors = d_cursors;
                    matrix.d_cdf_data = d_cdf_data;
                    matrix.d_ppf_data = d_ppf_data;
                    matrix.d_payload =  d_payload;
                }

                hipMemcpy(vec, v0, sizeof(int8_t)*len_v, hipMemcpyDeviceToDevice);

                hipEventRecord(start2);

                float v_delta = 1;

                // COMPUTE LOOP
                for (int k = 0; k<num_matrices; k++){
                    CompressedMatrix& matrix = encoded_matrices[k];

                    matrix.decompressAndMult(d_result, d_result32, vec, v_delta);
                    checkCUDAError("after decompressing matrix");

                    hipMemcpy(h_result,d_result, sizeof(int8_t)*rows,hipMemcpyDeviceToHost);

                    // to swap variables you need a third guy 'tmp'
                    tmp = vec;
                    vec = d_result;
                    d_result = tmp;


                    checkCUDAError("sizes misalign");
                    rows = matrix.rows;
                }
                hipDeviceSynchronize();



                checkCUDAError("Before Memcpy.");

                // copy 'vec' since we swapped it with d_result
                hipEventRecord(stop2);
                hipEventSynchronize(stop2);

                hipMemcpy(h_result, vec, sizeof(int8_t)* rows, hipMemcpyDeviceToHost);
                hipEventRecord(stop1);

                // freeing memory is not considered here
                hipEventSynchronize(stop1);
                uint32_t compute_hash = hash_int8_array(h_result, rows);
                if (compute_hash == result_hash){
                    printf("Hashes match! (%u)\n", compute_hash);
                }
                else{
                    printf("Hashes *don't* match!\n");
                    printf("[");
                    for (int i=0; i<rows; i++){
                        // printf("Result at index %d: %d\n", i, h_result[i]);
                        printf("%d,",  h_result[i]);
                    }
                    printf("]\n");
                }

                hipEventElapsedTime(&ms1, start1, stop1);
                hipEventElapsedTime(&ms2, start2, stop2);
                
                for (int k = 0; k<num_matrices; k++){
                    CompressedMatrix& matrix = encoded_matrices[k];
                    hipFree(matrix.d_cursors);
                    hipFree(matrix.d_cdf_data);
                    hipFree(matrix.d_ppf_data);
                    hipFree(matrix.d_payload);
                }

                float throughput = 1000.0 * num_matrices * rows * rows  / ms2;

                printf("Time with memcpy:    %f ms\n", ms1);
                printf("Time without memcpy: %f ms\n", ms2);
                std::cout<< "Throughput: " << std::scientific << throughput << " MAC/s" << std::endl;
            }

            for (int k = 0; k<num_matrices; k++){
                CompressedMatrix& matrix = encoded_matrices[k];
                hipHostFree(matrix.cursors);
                hipHostFree(matrix.cdf_data);
                hipHostFree(matrix.ppf_data);
                hipHostFree(matrix.payload);
            }
            // show result
            hipFree(d_result);
            hipFree(d_result32);
            hipFree(v0);
            hipFree(vec);
            delete[] h_v0;
            delete[] h_result;


            hipDeviceReset();
            checkCUDAError("End of program.");
        }
    }
    
    if (!opened) {
        std::cerr << "Error: Could not open a  from any of the given paths." << std::endl;
        return 1;
    }
    
    return 0;
}

void checkCUDAError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}
