#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdio>
#include <fstream>
#include <cstdint>
#include <cstring>
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>

#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>

#define MAX_BLOCKS 256
#define MAX_THREADS 256
#define WARP_SIZE 32

void checkCUDAError(const char* msg);

__inline__ __device__ int8_t find_r(uint8_t quantile, uint8_t* cdf, int G){

    for (int8_t r=G; r>0; r--){
        if (cdf[r-1] <= quantile){
            return r-1;
        }
    }
    return -1;
};

uint32_t hash_int8_array(int8_t* arr, int size)
{
    uint32_t hash = 0;

    for (size_t i = 0; i < size; i++)
    {
        hash = (hash >> 27) | (hash << 5); // Rotate left by 5 bits
        hash = (hash ^ *reinterpret_cast<const uint8_t *>(&arr[i])) * 0x27220A95;
    }

    return hash;
}
template<int Capacity>
struct ThreadQueue {
    uint16_t data[Capacity];
    int head;  // index for dequeue
    int tail;  // index for enqueue
    int size;
    int4* data4;

    __device__ void init() {
        head = 0;
        tail = 0;
        size = 0;
    }

    __device__ bool isEmpty() const {
        return size == 0;
    }

    __device__ bool isFull() const {
        return size > (Capacity-8);
    }

    __device__ bool enqueue(int4 val) {
        if (isFull()) return false;

        data4 = reinterpret_cast<int4*>(&data[tail]);
        data4[0] = val;
        tail = (tail + 8) % Capacity;
        size+=8;
        return true;
    }

    // Schnelles dequeue: direkter Zugriff, minimale Logik
    __device__ bool dequeue(uint16_t &out) {
        if (isEmpty()) return false;
        out = data[head];
        head = (head + 1) % Capacity;
        size--;
        return true;
    }
};

struct AbsValue {
    __host__ __device__
    float operator()(const int32_t& x) const {
        return abs(x);
    }
};

float absMaxWithThrustDevice(int32_t* d_input, int n) {
    thrust::device_ptr<int32_t> dev_ptr(d_input);

    return thrust::transform_reduce(
        dev_ptr, dev_ptr + n,
        AbsValue(),              // transform: fabs(x)
        0.0f,                    // init
        thrust::maximum<int32_t>() // reduce: max
    );
}

__global__ void normalizeAndRoundtoInt8(int32_t* res32, int8_t* res8, float scalar, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        int32_t a = res32[idx];
        float afl = static_cast<float>(a);

        a = __float2int_rn(afl/ scalar);
        res8[idx] = static_cast<int8_t>(a);
    }
}


class CompressedMatrix {
public:
    uint32_t rows, cols;
    float grid_spacing;
    int8_t min_value;
    uint8_t G;
    // host array pointers
    uint32_t* cursors;
    uint8_t* cdf_data;
    uint8_t* ppf_data;
    uint16_t* payload;
    uint32_t payload_size;

    // device array pointers
    uint32_t* d_cursors;
    uint8_t* d_cdf_data;
    uint8_t* d_ppf_data;
    uint16_t* d_payload;

    CompressedMatrix(uint32_t r, uint32_t c, float gs, uint32_t* cur,
                      int8_t minVal, uint8_t G, uint8_t* cdf, uint8_t* ppf, uint16_t* pay, uint32_t pay_size)
        : rows(r), cols(c), grid_spacing(gs), cursors(cur), min_value(minVal), G(G),
          cdf_data(cdf), ppf_data(ppf), payload(pay), payload_size(pay_size), d_cursors(nullptr), d_cdf_data(nullptr),
           d_ppf_data(nullptr), d_payload(nullptr){}

    __host__ static CompressedMatrix deserialize(std::ifstream& file) {
        uint32_t rows, cols;
        float grid_spacing;

        uint32_t payload_size;
        int8_t min_value;
        uint8_t G;
        file.read(reinterpret_cast<char*>(&rows), sizeof(rows));
        file.read(reinterpret_cast<char*>(&cols), sizeof(cols));
        file.read(reinterpret_cast<char*>(&grid_spacing), sizeof(float));
        
        // uint32_t* cursors = new uint32_t[rows];

        // non-pageable memory
        uint32_t* cursors;
        hipHostMalloc(&cursors, sizeof(uint32_t)*rows);

        file.read(reinterpret_cast<char*>(cursors), rows * sizeof(uint32_t));

        file.read(reinterpret_cast<char*>(&payload_size), sizeof(payload_size));

        file.read(reinterpret_cast<char*>(&min_value), sizeof(int8_t));
        file.read(reinterpret_cast<char*>(&G), sizeof(uint8_t));

        uint32_t cdf_len = G + 1;
        // uint8_t* cdf_data = new uint8_t[cdf_len];
        uint8_t* cdf_data;
        hipHostMalloc(&cdf_data, sizeof(uint8_t)*cdf_len);
        file.read(reinterpret_cast<char*>(cdf_data), cdf_len);
        
        if (cdf_len % 2 == 1) {
            file.seekg(1, std::ios::cur);
        }

        // uint8_t* ppf_data = new uint8_t[256];;
        uint8_t* ppf_data;
        hipHostMalloc(&ppf_data, sizeof(uint8_t)*256);
        file.read( reinterpret_cast<char*>(ppf_data), 256);
        
        // uint16_t* payload = new uint16_t[payload_size];
        __align__(16) uint16_t* payload;
        hipHostMalloc(&payload, sizeof(uint16_t)*payload_size);
        file.read(reinterpret_cast<char*>(payload), payload_size * sizeof(uint16_t));
        
        if (payload_size % 2 == 1) {
            file.seekg(2, std::ios::cur);
        }

        return CompressedMatrix(rows, cols, grid_spacing, cursors, min_value, G, cdf_data, ppf_data, payload, payload_size);
    }

    float decompressAndMult(int8_t* result, int32_t* d_result32, int8_t* vector, float v_delta);
};

__global__ void decmpressAndMultiply(int32_t* dst, int8_t* vec,
     uint32_t rows, uint32_t cols, float grid_spacing,
     const uint32_t* cursors, int8_t min_value, uint8_t G,
     const uint8_t* cdf_data, const uint8_t* ppf_data, uint16_t* payload, uint32_t payload_size
){
    unsigned int threadNo = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int tId = threadIdx.x;
    unsigned int lane = tId % 32;
    // one warpPerRow means warpId == row
    unsigned int warpId = threadNo / 32;
    // unsigned int bId = blockIdx.x;
    unsigned int blockSize = blockDim.x;
    
    
    uint32_t cursor;
    uint32_t head;
    uint8_t quantile;
    uint8_t r;
    int32_t w;
    uint8_t prob;
    uint16_t word;
    int8_t* bytes;

    __shared__ int8_t shared_vec[4096]; // TODO NOT HARDCODE THIS NUMBER
    extern __shared__ uint32_t cdf[]; // store cdf in shared memory
    __shared__ uint8_t ppf[256];





    int4* shared_vec4 = reinterpret_cast<int4*>(shared_vec);
    int4* vec4 = reinterpret_cast<int4*>(vec);
    for (int j = tId; j <(cols/16); j+=blockSize ){
        shared_vec4[j] =  vec4[j];
    }
    // int32_t* shared32 = reinterpret_cast<int32_t*>(shared_vec);

    int32_t res = 0;

    // loads cdf & ppf into shared memory 
    for (int j = tId; j <G+1; j+=blockSize ){
        cdf[j] = cdf_data[j];
    }
    for (int j=tId; j< 256; j+=blockSize){
        ppf[j] = ppf_data[j];
    }

    __syncthreads();
    
    ThreadQueue<16> q;
    q.init();
    cursor = cursors[warpId];
    head = payload[cursor+lane*8] << 16 | payload[cursor+lane*8+1];
    int4* payload4 = reinterpret_cast<int4*>(&payload[cursor]);
    int cursor4 = lane;

    int shared_vec_cursor = lane;
    int4 v4;
    bool remmi = q.enqueue(payload4[cursor4]);
    // dequeue 2 words since first two words are used for the coder head
    q.dequeue(word);
    q.dequeue(word);
    cursor4+=32;

    int8_t w0, w1, w2, w3;
    for (int j = 0; j < cols/WARP_SIZE; j++){
            // does not diverege across warp
            if (j%16==0){
                v4 = shared_vec4[shared_vec_cursor];
                shared_vec_cursor+=WARP_SIZE;
                bytes = reinterpret_cast<int8_t*>(&v4);
            }


            quantile = head & ((1<<8)-1); // take first 8 bits of head as quantile

            r = ppf[quantile];
            // r = find_r(quantile, cdf, G);


            w0 = min_value + r;

            // __dp4a(srcA, srcB,c); 
            // see https://docs.nvidia.com/cuda/cuda-math-api/cuda_math_api/group__CUDA__MATH__INTRINSIC__INT.html#group__cuda__math__intrinsic__int_1ga933213059df6da2de206771f145ac2f8



            res += w0 * bytes[j%16]; // perform scalar addition

            prob = (cdf[r+1] - cdf[r]) & 0xFF; // modulo 2**8 to ensure it fits in a uint8
            head = (head >> 8) * prob  + (quantile -cdf[r]);
            // if (warpId == 136 && lane==16){
            //     // printf("decoded %d,   col: %d, lane: %d \n", w0, j, lane);
            //     printf("%d, \t %d \n ",  bytes[j%16],w0);
            // }

            bool this_lane_needs_refill = q.isEmpty();
            unsigned int any_result = __any_sync(0xFFFFFFFF, this_lane_needs_refill);
            // does not diverege across warp
            if (any_result){
                q.enqueue(payload4[cursor4]);
                // XOR
                // if (!q.enqueue(payload4[cursor4])){
                //     printf("Wir haben kein remmydemmi!!\n");
                // };
                cursor4+=32;
            }
            
            // diverege across warp!
            if (head < (1<<16)){
                q.dequeue(word);
                // XOR
                // if(!q.dequeue(word)){
                //     printf("Wir haben kein remmydemmi!!\n");
                // }
                head = head<<16 | word;
            }
        
        }   
        __syncwarp();

        // if (warpId==136) printf("lane: %d, %d\n", lane, res);

        // Warp reduction (sum partials across lanes)
        for (int offset = WARP_SIZE/2; offset > 0; offset >>= 1) {
            res += __shfl_down_sync(0xffffffff, res, offset);
        }
        __syncwarp();


        // Lane 0 writes the row result
        if (lane == 0){
            // printf("warpId: %d, res: %d \n", warpId, res);
            dst[warpId] = res;
        }

        // dst[threadNo] = res;
}
float CompressedMatrix::decompressAndMult(int8_t* d_result8, int32_t* d_result32, int8_t* vector, float v_delta){

    // dim3 blockGrid(MAX_BLOCKS);
    // dim3 threadBlock(MAX_THREADS);
    
    int rows = this->rows;
    int cols = this->cols;
    
    int warpsPerBlock = 4;
    dim3 blockGrid( (rows + warpsPerBlock - 1) / warpsPerBlock );
    dim3 threadBlock(warpsPerBlock * 32);
    

    decmpressAndMultiply<<<blockGrid, threadBlock, (G+1)*sizeof(int32_t)>>>(d_result32, vector,
        this->rows, this->cols, this->grid_spacing,
        this-> d_cursors, this->min_value, this->G,
        this->d_cdf_data,this->d_ppf_data, this->d_payload, this->payload_size);


    float abs_max = absMaxWithThrustDevice(d_result32, this->rows);
    v_delta = abs_max / 127;
    
    int blocks = (rows+ MAX_THREADS - 1) / MAX_THREADS;
    normalizeAndRoundtoInt8<<< blocks,MAX_THREADS>>>
    (d_result32, d_result8, v_delta, this->rows);

    checkCUDAError("after kernel");
    return v_delta;
}


int main() {
    // Open the binary file
    std::vector<std::string> filepaths = {
        "/home/wildug/RSP/myKernel/matvansWarpPerRow/compressed_matrices_4096_5bit.bin",
    };

    bool opened = false;
    std::ifstream file;
    for (const auto& path : filepaths) {
        file.open(path);
        if (file.is_open()) {
            opened = true;
            break;
        }
    }
    
    if (!opened) {
        std::cerr << "Error: Could not open file from any of the given paths." << std::endl;
        return 1;
    }
    // for timing
    // time including memcpy
    float ms1 = 0;
    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);

    // time using on
    float ms2 = 0;
    hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);


    uint32_t num_matrices, result_hash, max_word_count, len_v;
    int8_t* v0;
    int8_t* vec;
    file.read(reinterpret_cast<char*>(&num_matrices), sizeof(num_matrices));
    file.read(reinterpret_cast<char*>(&result_hash), sizeof(result_hash));
    file.read(reinterpret_cast<char*>(&max_word_count), sizeof(max_word_count));
    file.read(reinterpret_cast<char*>(&len_v), sizeof(len_v));


    int8_t* h_v0 = new int8_t[len_v];
    file.read(reinterpret_cast<char*>(h_v0), len_v*sizeof(uint8_t));

    hipMalloc(&v0, sizeof(int8_t)*len_v);
    hipMemcpy(v0, h_v0,sizeof(int8_t)*len_v,  hipMemcpyHostToDevice);

    hipMalloc(&vec,sizeof(int8_t)*len_v);

    std::cout << "Number of matrices: " << num_matrices << std::endl;
    std::cout << "Max word-count: " << max_word_count << std::endl;
    std::cout << "len_v: " << len_v << std::endl;



    // maybe first read all files and then do the mat vec operation
    int8_t* d_result;
    int8_t* h_result;
    int rows =len_v;
    int8_t* tmp;
    std::vector<CompressedMatrix> encoded_matrices;

    for (int k = 0; k<num_matrices; k++){
        CompressedMatrix matrix = CompressedMatrix::deserialize(file);
        encoded_matrices.push_back(std::move(matrix));
    }


    file.close();

    h_result = new int8_t[rows];
    

    int max_rows = 0;
    for (int k = 0; k<num_matrices; k++){
        if (max_rows < encoded_matrices[k].rows)              
            max_rows = encoded_matrices[k].rows;
    }


    hipMalloc(&d_result, sizeof(int8_t)* max_rows); 

    int32_t* d_result32;
    hipMalloc(&d_result32, sizeof(int32_t)* max_rows); 
    
    // MEMCPY LOOP, move hipEventRecord above or below

    int NUM_ITERATIONS = 10;
    for (int l=0; l< NUM_ITERATIONS; l++){ // outer loop for benchmarking

        hipEventRecord(start1);
        for (int k = 0; k<num_matrices; k++){
            CompressedMatrix& matrix = encoded_matrices[k];
            uint32_t* d_cursors;
            uint8_t* d_cdf_data;
            uint8_t* d_ppf_data;
            __align__(16) uint16_t* d_payload;

            // malloc
            checkCUDAError("before Malloc");
            hipMalloc(&d_cursors, sizeof(uint32_t)* matrix.rows);
            hipMalloc(&d_cdf_data, sizeof(uint8_t)*(matrix.G +1));
            hipMalloc(&d_ppf_data, 256*sizeof(uint8_t));
            hipMalloc(&d_payload, matrix.payload_size * sizeof(uint16_t));

            checkCUDAError("after Malloc");

            // memcpy
            hipMemcpy(d_cursors, matrix.cursors, sizeof(uint32_t)*matrix.rows, hipMemcpyHostToDevice);
            hipMemcpy(d_cdf_data, matrix.cdf_data,sizeof(uint8_t)*(matrix.G +1), hipMemcpyHostToDevice);
            hipMemcpy(d_ppf_data, matrix.ppf_data,sizeof(uint8_t)*256, hipMemcpyHostToDevice);
            hipMemcpy(d_payload, matrix.payload, matrix.payload_size * sizeof(uint16_t), hipMemcpyHostToDevice);

            checkCUDAError("after Memcpy");


            // set the *device* pointer as object attribute
            matrix.d_cursors = d_cursors;
            matrix.d_cdf_data = d_cdf_data;
            matrix.d_ppf_data = d_ppf_data;
            matrix.d_payload =  d_payload;
        }

        hipMemcpy(vec, v0, sizeof(int8_t)*len_v, hipMemcpyDeviceToDevice);

        hipEventRecord(start2);

        float v_delta = 1;

        // COMPUTE LOOP
        for (int k = 0; k<num_matrices; k++){
            CompressedMatrix& matrix = encoded_matrices[k];

            matrix.decompressAndMult(d_result, d_result32, vec, v_delta);
            checkCUDAError("after decompressing matrix");

            // print intermediate hashes
            hipMemcpy(h_result,d_result, sizeof(int8_t)*rows,hipMemcpyDeviceToHost);
            // printf("[");
            // for (int i=0; i<rows;i++){
            //     printf("%d, ",h_result[i]);
            // }
            // printf("]\n");

            // uint hash = hash_int8_array(h_result,rows);
            // printf("Hash my darling: %d\n", hash);


            // to swap variables you need a third guy 'tmp'
            tmp = vec;
            vec = d_result;
            d_result = tmp;


            checkCUDAError("sizes misalign");
            rows = matrix.rows;
        }
        hipDeviceSynchronize();



        checkCUDAError("Before Memcpy.");

        // copy 'vec' since we swapped it with d_result
        hipEventRecord(stop2);
        hipEventSynchronize(stop2);

        hipMemcpy(h_result, vec, sizeof(int8_t)* rows, hipMemcpyDeviceToHost);
        hipEventRecord(stop1);

        // freeing memory is not considered here
        hipEventSynchronize(stop1);
        uint32_t compute_hash = hash_int8_array(h_result, rows);
        if (compute_hash == result_hash){
            printf("Hashes match! (%d)\n", compute_hash);
        }
        else{
            printf("Hashes *don't* match!\n");
            printf("[");
            for (int i=0; i<rows; i++){
                // printf("Result at index %d: %d\n", i, h_result[i]);
                printf("%d,",  h_result[i]);
            }
            printf("]\n");
        }

        hipEventElapsedTime(&ms1, start1, stop1);
        hipEventElapsedTime(&ms2, start2, stop2);
        
        for (int k = 0; k<num_matrices; k++){
            CompressedMatrix& matrix = encoded_matrices[k];
            hipFree(matrix.d_cursors);
            hipFree(matrix.d_cdf_data);
            hipFree(matrix.d_ppf_data);
            hipFree(matrix.d_payload);
        }


        printf("Time with memcpy:    %f ms\n", ms1);
        printf("Time without memcpy: %f ms\n", ms2);
    }

    for (int k = 0; k<num_matrices; k++){
        CompressedMatrix& matrix = encoded_matrices[k];
        hipHostFree(matrix.cursors);
        hipHostFree(matrix.cdf_data);
        hipHostFree(matrix.ppf_data);
        hipHostFree(matrix.payload);
    }
    // show result
    hipFree(d_result);
    hipFree(d_result32);
    hipFree(v0);
    hipFree(vec);
    delete[] h_v0;
    delete[] h_result;


    hipDeviceReset();
    checkCUDAError("End of program.");
    
    return 0;
}

void checkCUDAError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}
