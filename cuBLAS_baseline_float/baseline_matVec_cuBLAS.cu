#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdint>
#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <fstream>
#include <vector>

#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <cmath>

#define EPSILON 1e-5 
#define MAX_BLOCKS 256
#define MAX_THREADS 256

#define VERBOSE

bool validateResults(float* hostMat, float* hostVec, float* hostResVec, int w, int h);
void checkCUDAError(const char* msg);

struct AbsValue {
    __host__ __device__
    float operator()(const float& x) const {
        return fabsf(x);
    }
};

uint32_t hash_int8_array(int8_t* arr, int size)
{
    uint32_t hash = 0;

    for (size_t i = 0; i < size; i++)
    {
        hash = (hash >> 27) | (hash << 5); // Rotate left by 5 bits
        hash = (hash ^ *reinterpret_cast<const uint8_t *>(&arr[i])) * 0x27220A95;
    }

    return hash;
}

float absMaxWithThrustDevice(float* d_input, int n) {
    thrust::device_ptr<float> dev_ptr(d_input);

    return thrust::transform_reduce(
        dev_ptr, dev_ptr + n,
        AbsValue(),              // transform: fabs(x)
        0.0f,                    // init
        thrust::maximum<float>() // reduce: max
    );
}

__global__ void normalizeAndRound(float* vec, float scalar, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        vec[idx] = roundf(vec[idx]/ scalar);
    }
}


class Matrix{
public:
    uint32_t rows;
    uint32_t cols;
    float w_delta;
    float* data;
    float* d_data;

    Matrix(uint32_t r, uint32_t c, float w_delta, float* d) : rows(r), cols(c), w_delta(w_delta), data(d) {}

    __host__ static Matrix deserialize(std::ifstream& file){
        uint32_t rows, cols;
        float grid_spacing;

        file.read(reinterpret_cast<char*>(&rows), sizeof(rows));
        file.read(reinterpret_cast<char*>(&cols), sizeof(cols));

        file.read(reinterpret_cast<char*>(&grid_spacing), sizeof(float));
        
        int8_t*  int_data = new int8_t[rows*cols];
        float* data; 
        printf("rows: %d, cols: %d\n",rows, cols);
        hipHostAlloc(&data, rows*cols*sizeof(float), hipHostMallocDefault);
        file.read(reinterpret_cast<char*>(int_data), rows*cols*sizeof(int8_t));

        
        // Casting to float here, watchout!
        for (int i = 0; i < rows*cols; ++i) {
                data[i] = static_cast<float>(int_data[i]);
        }

        if ((cols * rows) % 2 == 1){
            file.seekg(1, std::ios::cur);
            printf("Odd number of Matrix elements: skipped padding.");
        }

        delete [] int_data;

        return Matrix(rows, cols, grid_spacing, data);
    }
    
    float mult(hipblasHandle_t handle, float* result, float* vector, float v_delta);
};

float Matrix::mult(hipblasHandle_t handle, float* result, float* vector, float v_delta){
    // returns float value 

    float alpha = this->w_delta * v_delta;  // Scale of vector and matrix quantization
    float beta = 0.0f;   // Scalar multiplier for the initial value of y (should be 0 if we're just doing the product)
    int rows = this->rows;         // Leading dimension of matrix A
    int incx = 1;        // Increment for vector x
    int incy = 1;        // Increment for vector y
    int num_elems = this->cols*this->rows;



    hipblasStatus_t stat;
    checkCUDAError("Before Sgemv");

    stat = hipblasSgemv(handle, HIPBLAS_OP_T, rows, this->cols, &alpha, this->d_data, rows, vector, incx, &beta, result, incy);
    
    checkCUDAError("after Sgemv");
    float abs_max = absMaxWithThrustDevice(result, this->rows);

    v_delta = abs_max / 127;

    int blocks = (rows+ MAX_THREADS - 1) / MAX_THREADS;

    normalizeAndRound<<< blocks,MAX_THREADS>>>    (result, v_delta, rows);
    checkCUDAError("after normalizeAndRound");


    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("ERROR HIPBLAS_STATUS_SUCCESS)");
    }

    return v_delta;
}
int main(int argc, char* argv[]) {

    std::string filename = "/home/wildug/RSP/myKernel/raw-matrices_4096.bin";
    std::ifstream file(filename, std::ios::binary);
    if (!file) {
        std::cerr << "Error: Could not open file" << std::endl;
        return 1;
    }


    // Initialize cuBLAS handle
    float ms1 = 0;
    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);

    // time using on
    float ms2 = 0;
    hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);

    hipEventCreate(&start1);
    hipEventCreate(&start2);
    hipEventCreate(&stop1);
    hipEventCreate(&stop2);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    uint32_t num_matrices, len_v, result_hash;
    int8_t* int_vec;

    file.read(reinterpret_cast<char*>(&num_matrices), sizeof(num_matrices));
    file.read(reinterpret_cast<char*>(&result_hash), sizeof(result_hash));
    file.read(reinterpret_cast<char*>(&len_v), sizeof(len_v));

    int_vec = new int8_t[len_v];
    std::cout << "Number of matrices: " << num_matrices << std::endl;
    std::cout << "len_v: " << len_v << std::endl;
    file.read(reinterpret_cast<char*>(int_vec), len_v*sizeof(uint8_t));

    float* h_vec = new float[len_v];
    float* vec;
    float* v0;


    for (int i = 0; i < len_v; ++i) {
            h_vec[i] = static_cast<float>(int_vec[i]);
            // printf("%f, ",h_vec[i]);
    }
    printf("end\n");
    hipMalloc(&vec, sizeof(float)*len_v);
    hipMalloc(&v0, sizeof(float)*len_v);
    hipMemcpy(v0, h_vec, sizeof(float)*len_v,hipMemcpyHostToDevice);

    checkCUDAError("after Reading");

    std::vector<Matrix> matrices;
    for (int k = 0; k<num_matrices; k++){
        printf("%d\n",k);
        Matrix matrix = Matrix::deserialize(file);
        matrices.push_back(std::move(matrix));
    }
    file.close();

    for (int l=0; l< 20; l++){ // outer loop for benchmarking
        hipEventRecord(start1);
        // copy initial array to vec
        hipMemcpy(vec, v0, sizeof(float)*len_v,hipMemcpyDeviceToDevice);

        for (int k = 0; k<num_matrices; k++){
            Matrix& matrix = matrices[k];
            float* d_data;
            int num_elems = matrix.rows*matrix.cols;

            checkCUDAError("before Malloc");
            hipMalloc((void**)&d_data, sizeof(float)*num_elems);
            checkCUDAError("after Malloc");
            hipMemcpy(d_data, matrix.data,sizeof(float)*num_elems, hipMemcpyHostToDevice);
            checkCUDAError("after Memcpy");


            matrix.d_data = d_data;
        }


        checkCUDAError("after loop");
        int max_rows = 0;
        for (const auto& matrix : matrices) {
            if (matrix.rows > max_rows)
                max_rows = matrix.rows;
        }


        float* d_result;
        float* blob;
        hipMalloc(&d_result, sizeof(float)*max_rows);

        checkCUDAError("after allocating d_result");
        int rows;

        hipEventRecord(start2);
        int v_delta = 1; // scaling factor of v starts with 1

        for (int k = 0; k<num_matrices; k++){
            Matrix matrix = matrices[k];
            rows = matrix.rows;
            v_delta = matrix.mult(handle, d_result, vec, v_delta);

            checkCUDAError("after multiplying matrix");
            // to swap variables you need a third guy 'blob'
            blob = vec;
            vec = d_result;
            d_result = blob;
        }
        hipEventRecord(stop2);
        hipEventSynchronize(stop2);
        hipMemcpy(h_vec, vec, sizeof(float)*rows, hipMemcpyDeviceToHost);

        hipEventRecord(stop1);
        hipEventSynchronize(stop1);


        hipEventElapsedTime(&ms1, start1, stop1);
        hipEventElapsedTime(&ms2, start2, stop2);
        
        for (int k = 0; k<num_matrices; k++){
            Matrix matrix = matrices[k];
            hipFree(matrix.d_data);
        }

        printf("Time with memcpy:    %f ms\n", ms1);
        printf("Time without memcpy: %f ms\n", ms2);
    }

    hipFree(vec);



    // Output result
    std::cout << "Result vector y: ";
    for (int i = 0; i < len_v; i++) {
        std::cout << h_vec[i] << ", ";
    }
    std::cout << std::endl;



    hipblasDestroy(handle);

    return 0;
}

bool validateResults(float* hostMat, float* hostVec, float* gpuResult, int w, int h) {

    float* result = new float[h];

    for (int i = 0; i < h; ++i) {
        result[i] = 0.0f;
        for (int j = 0; j < w; ++j) {
            result[i] += hostMat[i * w + j] * hostVec[j];
        }
    }
    for (int i = 0; i < h; ++i) {
        if (fabs(result[i] - gpuResult[i]) > EPSILON) {
            printf("Mismatch at index %d: CPU=%f, GPU=%f\n", i, result[i], gpuResult[i]);
            return false;
        }
    }
    delete [] result;
    return true;
}

void checkCUDAError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}